/*
   call cublas<S>dot function and compare CPU and GPU time
   Sinan SHI
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>
//#include <helper_cuda.h>
#define N 100000000

int cpu_dot(const float *x, const float *y, float *z){
    int i;
    z[0]=0;
    for(i=0;i<N;i++){
        z[0]=z[0]+x[i]*y[i];
    }
    return(0);
}



int check(hipblasStatus_t status){
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
    
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    
    }
    return(0);
}
    

/*Main function*/
int main(int argc, char **argv)
{



    clock_t start,diff;
    hipblasHandle_t handle;
    hipblasStatus_t status;
    float *x;
    float *y;
    float *z,*zg;
    float *d_x, *d_y, *d_z;
    int i;

    status = hipblasCreate(&handle);
    x=(float *)malloc(N * sizeof(x[0]));
    y=(float *)malloc(N * sizeof(y[0]));
    z=(float *)malloc(sizeof(float));
    zg=(float *)malloc(  sizeof(float));


    hipMalloc((void **)&d_x, N * sizeof(float));
    hipMalloc((void **)&d_y, N * sizeof(float));
    hipMalloc((void **)&d_z, sizeof(float));


    /*initialise*/
    for(i=0;i<N;i++){
        x[i]=0.001;
        y[i]=0.001;
        zg[0]=-9999;
        z[0]=-9999;
    }

    
    /*CPU dot product*/
    start=clock();
    cpu_dot(x,y,z);
    diff=clock()-start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("Time taken (CPU) %d seconds %d milliseconds\n", msec/1000, msec%1000);
    


    /*GPU dot product*/
    start=clock();
    hipblasSetVector(N,sizeof(float),x,1,d_x,1);
    hipblasSetVector(N,sizeof(float),y,1,d_y,1);
    check(hipblasSdot(handle,N,d_x,1,d_y,1,zg));
    diff=clock()-start;
    msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("Time taken (GPU) %d seconds %d milliseconds\n", msec/1000, msec%1000);
    

    /*check result*/
    printf("%f - %f\n",zg[0],z[0]);





    free(x);
    free(y);
    free(z);
    free(zg);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
}
